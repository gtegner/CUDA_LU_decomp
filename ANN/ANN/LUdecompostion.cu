#include "hip/hip_runtime.h"

#include <hip/device_functions.h>


#include <sm_30_intrinsics.h>

#ifndef __CUDACC__
#define __CUDACC__
#endif // !__CUDACC__

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <float.h>
#include <algorithm> 
#include <iostream>
#include <iomanip>
#include <hiprand/hiprand.h>
#include <ctime>

#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
int __shfl_down(double var, unsigned int delta, int width = warpSize);
int __shfl_down(int var, unsigned int delta, int width = warpSize);
#endif

using namespace std;

ostream& operator<<(ostream& out, pair<int, double**> mat)
{
	int m = mat.first;
	double** arr = mat.second;

	cout << setprecision(5) << fixed << endl;
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < m; j++) {
			cout << arr[i][j] << (j + 1 < m ? ",\t" : "\n");
		}
	}
	return out;
}

#define check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define r_check(ans) { curandAssert((ans)); }
inline void curandAssert(hiprandStatus_t code, bool abort = true)
{
	if (code != HIPRAND_STATUS_SUCCESS)
	{
		if (abort) exit(code);
	}
}

//__device__ inline
//double __shfl_down(double var, unsigned int srcLane, int width = 32) {
//	int2 a = *reinterpret_cast<int2*>(&var);
//	a.x = __shfl_down(a.x, srcLane, width);
//	a.y = __shfl_down(a.y, srcLane, width);
//	return *reinterpret_cast<double*>(&a);
//}

__inline__ __device__ void max_ind_warp(int& max_i, double& max_v ) {
	#pragma unroll
	for (int offset = 32 / 2; offset > 0; offset /= 2) {
		double cand_v = __shfl_down(max_v, offset);
		int cand_i = __shfl_down(max_i, offset);
		if (abs(cand_v) > abs(max_v)) {
			max_v = cand_v;
			max_i = cand_i;
		}
	}
}

__inline__ __device__ void max_ind_block(int& max_i, double& max_v)
{
	__shared__ double max_vs[32];
	__shared__ int max_is[32];

	max_ind_warp(max_i, max_v);

	//  printf("Thread %i: %i %f \n", threadIdx.x, max_i, max_v);

	int lane = threadIdx.x % warpSize;
	int warpID = threadIdx.x / warpSize;

	if (lane == 0) {
		max_vs[warpID] = max_v;
		max_is[warpID] = max_i;
	}

	__syncthreads();

	max_v = (threadIdx.x < 32) ? max_vs[lane] : 0.0;
	max_i = (threadIdx.x < 32) ? max_is[lane] : 0;
	
	// printf("Thread %i: %i %f \n", threadIdx.x, max_i, max_v);

	max_ind_warp(max_i, max_v);

	// printf("Thread %i: %i %f \n", threadIdx.x, max_i, max_v);

}

// Allways call with only one block and 1024 threads!
__global__ void maxColumn(double ** __restrict__ mat, const int c, const int N, int* __restrict__ P)
{
	int numThreads = blockDim.x * gridDim.x;
	int I = threadIdx.x + blockIdx.x * blockDim.x;

	int max_i = I;
	double max_v = 0.0;
	for (int i = c + I; i < N; i += numThreads)
	{
		double cand_v = mat[i][c];
		if (abs(cand_v) >= abs(max_v)) {
			max_v = cand_v;
			max_i = i;
		}
	}

	max_ind_block(max_i, max_v);

	if (threadIdx.x == 0) {
		// printf("%i %f \n", max_i, max_v);
		double* tmp = mat[max_i];
		mat[max_i] = mat[c];
		mat[c] = tmp;

		// printf("max_i %i, P[c] %i, P[max_i] %i \n", max_i, P[c], P[max_i]);
		int itmp = P[c];
		P[c] = P[max_i];
		P[max_i] = itmp;
	}
}

__global__ void compute_L_column (double ** __restrict__ mat, const int col, const int N) {
	double diag = mat[col][col];

	int numThreads = blockDim.x * gridDim.x;
	int I = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = I + col + 1; i < N; i += numThreads) {
		mat[i][col] /= diag;
	}
}

__global__ void reduce(double ** __restrict__ mat, const int n, const int N) {

	__shared__ double actv[16];
	__shared__ double fact[16];

	//int numThreadsX = blockDim.x * gridDim.x;
	//int numThreadsY = blockDim.y * gridDim.y;

	int I = threadIdx.x + blockIdx.x * blockDim.x;
	int J = threadIdx.y + blockIdx.y * blockDim.y;

	if (I + n + 1 < N && threadIdx.y == 0) {
		actv[threadIdx.x] = mat[n][I + n + 1];
	}

	if (J + n + 1 < N && threadIdx.x == 0) {
		fact[threadIdx.y] = mat[J + n + 1][n];
	}

	__syncthreads();

	if (I + n + 1 < N && J + n + 1 < N) {
		mat[J + n + 1][I + n + 1] -= actv[threadIdx.x] * fact[threadIdx.y];
	}

}

__global__ void L_substitution(double ** __restrict__ mat, double* __restrict__ b, const int N) {
	extern __shared__ double _b[];

	int numThreads = blockDim.x * gridDim.x;
	int I = threadIdx.x + blockIdx.x * blockDim.x;

	//for (int i = I; i < N; i += numThreads) {
	//	printf("%i %f, ", i, b[i]);
	//	_b[i] = b[i];
	//}

	// on diagonal there are ones which are however not saved
	for (int j = 0; j < N; j++) {
		double b_j = _b[j];
		for (int i = j + 1 + I; i < N; i += numThreads) {
			_b[i] -= mat[i][j] * b_j;
		}
	}

	for (int i = I; i < N; i += numThreads) {
		// printf("%i %f, ",i, _b[i]);
		b[i] = _b[i];
	}
}

__global__ void U_substitution(double ** __restrict__ mat, double* __restrict__ b, const int N) {
	extern __shared__ double _b[];

	int numThreads = blockDim.x * gridDim.x;
	int I = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = I; i < N; i += numThreads) {
		_b[i] = b[i];
	}

	for (int j = N - 1; j >= 0; j--) {
		if ((j % numThreads) == I) {
			_b[j] /= mat[j][j];
		}
		__syncthreads();
		double b_j = _b[j];
		for (int i = I; i < j; i += numThreads) {
			_b[i] -= mat[i][j] * b_j;
		}
	}

	for (int i = I; i < N; i += numThreads) {
		b[i] = _b[i];
	}
}

// Run with 1 Block 1024 Threads only please!
__global__ void permute(double* d_b, const int* P, int const N) {

	extern __shared__ double _b[];

	int numThreads = blockDim.x * gridDim.x;
	int I = threadIdx.x + blockIdx.x * blockDim.x;

	//for (int i = I; i < N; i += numThreads) {
	//	printf("%i %f ", i, d_b[i]);
	//}
	// printf("\n ");

	for (int i = I; i < N; i += numThreads) {
		// printf("I=%i, i=%i, P[i]=%i, %f \n", I, i, P[i], d_b[P[i]]);
		_b[i] = d_b[P[i]];
	}

	for (int i = I; i < N; i += numThreads) {
		d_b[i] = _b[i];
	}
}

void LU_decompostion(double** d_mat, const int N, int* &P) {

	//check(cudaMallocManaged(&d_mat, N * sizeof(double*)));
	//for (int i = 0; i < N; i++) {
	//	// cudaMalloc
	//	check(cudaMallocManaged(&(d_mat[i]), N * sizeof(double)));
	//}

	//for (int i = 0; i < N; i++) {
	//	check(cudaMemcpyAsync(&(d_mat[i]), &(mat[i]), N, cudaMemcpyHostToDevice));
	//}


	check(hipMallocManaged(&P, N * sizeof(int)));
	for (int i = 0; i < N; i++) {
		P[i] = i;
	}

	hipDeviceSynchronize();

	unsigned int num_threads = min(256, N);
	unsigned int num_blocks = (N + num_threads - 1) / num_threads;
	dim3 threadsGrid = dim3(16, 16);

	for (int i = 0; i < N; i++) {
		maxColumn<<<1, 1024>>>(d_mat, i, N, P);
		compute_L_column<<<num_blocks, num_threads>>>(d_mat, i, N);
		
		int dim = (N + 16 - 1) / 16;
		dim3 blockgrid = dim3(dim, dim);
		reduce<<<blockgrid, threadsGrid>>>(d_mat, i, N);

		//cudaDeviceSynchronize();
		//auto Mat = make_pair(N, d_mat);
		//cout << endl << Mat << endl;
		//for (int i = 0; i < N; i++) {
		//	printf("%i, ", P[i]);
		//} printf("\n");
	}
}

// deconstructs b during execution
// assume d_mat and d_P are device pointer
void inline solve_LU(double** d_mat, int* d_P, double* b, const int N) {
	double* d_b;
	check(hipMallocManaged(&d_b, N * sizeof(double)));
	check(hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyDefault));

	unsigned int num_threads = min(256, N);
	unsigned int num_blocks = (N + num_threads - 1) / num_threads;

	permute<<<1, 1024, N * sizeof(double)>>>(d_b, d_P, N);
	hipDeviceSynchronize();

	L_substitution<<<num_blocks, num_threads, N * sizeof(double) >>>(d_mat, d_b, N);
	U_substitution<<<num_blocks, num_threads, N * sizeof(double) >>>(d_mat, d_b, N);
	hipDeviceSynchronize();

	check(hipMemcpy(b, d_b, N * sizeof(double), hipMemcpyDefault));
}

void test_maxColumn(double ** __restrict__ mat, const int c, const int N, int* P) {

	maxColumn<<<1,8>>>(mat, c, N, P);

	hipDeviceSynchronize();

	for(int i = 0; i < N; i++){
		cout << P[i] << ", ";
	}

}

void test_L_column(double ** __restrict__ mat, const int c, const int N) {

	compute_L_column<<<1, 8 >>>(mat, c, N);

	hipDeviceSynchronize();
	auto Mat = make_pair(N, mat);
	cout << endl << Mat << endl;

}

void test_reduce(double ** __restrict__ d_mat, const int c, const int N) {
	int* P;
	check(hipMallocManaged(&P, N * sizeof(int)));
	for (int i = 0; i < N; i++) {
		P[i] = i;
	}

	test_maxColumn(d_mat, c, N, P);
	test_L_column(d_mat, c, N);

	dim3 threadsGrid = dim3(4, 4);
	int dim = (N + 4 - 1) / 4;
	dim3 blockgrid = dim3(dim, dim);

	reduce<<<blockgrid, threadsGrid >>>(d_mat, c, N);

	hipDeviceSynchronize();
	auto Mat = make_pair(N, d_mat);
	cout << endl << Mat << endl;
}

void test_LU_decomp(double ** __restrict__ d_mat, int N) {
	int* P;
	LU_decompostion(d_mat, N, P);

	hipDeviceSynchronize();
	auto Mat = make_pair(N, d_mat);
	cout << endl << Mat << endl;

	double* b = (double*) calloc(sizeof(double), N);
	for (int i = 0; i < N; i++) {
		b[i] = i;
	}

	hipDeviceSynchronize();

	solve_LU(d_mat, P, b, N);
	hipDeviceSynchronize();

	for (int i = 0; i < N; i++) {
		cout << b[i] << ", ";
	}

}

void run_testsuit() {
	const int N = 5;

	//double mat_[N][N] = {{ 1,2,3 }, 
	//					 { 4,4,6 },
	//					 { 1,2,9 }};


	double mat_[N][N] = { { 17,24, 1, 8,15 },
					      { 23, 5, 7,14,16 },
						  {  4, 6,13,20,22 },
						  { 10,12,19,21, 3 },
						  { 11,18,25, 2, 9 } };
	double** mat;
	check(hipMallocManaged(&mat, sizeof(double*) * N));
	for (int i = 0; i < N; i++) {
		check(hipMallocManaged(&(mat[i]), sizeof(double) * N))
			// check(cudaMemcpy(mat[i], mat_[i], 3, cudaMemcpyHostToDevice));
	}
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			mat[i][j] = mat_[i][j];
		}
	}

	// test_maxColumn(mat, 2, N);
	// test_L_column(mat, 0, N);
	// test_reduce(mat, 0, N);
	// test_LU_decomp(mat, N);
}

void run_perfomance_test(const int N) {
	double** mat;
	check(hipMallocManaged(&mat, sizeof(double*) * N));

	hiprandGenerator_t gen;
	r_check(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	unsigned long int seed = static_cast<unsigned long int> (time(NULL));
	r_check(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

	for (int i = 0; i < N; i++) {
		check(hipMallocManaged(&(mat[i]), sizeof(double) * N))
		r_check(hiprandGenerateUniformDouble(gen, mat[i], N));
		hipDeviceSynchronize();
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int* P;
	hipEventRecord(start);

	LU_decompostion(mat, N, P);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	cout << "Time " << milliseconds << " for matrix of size " << N << "x" << N << endl;

	// FREE
	r_check(hiprandDestroyGenerator(gen));

	for (int i = 0; i < N; i++) {
		check(hipFree(mat[i]));
	}
	check(hipFree(mat));

	return;
}


int main() {
	
	//run_perfomance_test(100);
	//run_perfomance_test(300);
	//run_perfomance_test(900);
	//run_perfomance_test(900 * 2);
	//run_perfomance_test(900 * 3);
	//run_perfomance_test(900 * 4);
	//run_perfomance_test(900 * 5);
	//run_perfomance_test(900 * 6);

	run_perfomance_test(50);
	run_perfomance_test(50 * 2);
	run_perfomance_test(50 * 3);
	run_perfomance_test(50 * 4);
	run_perfomance_test(50 * 5);
	run_perfomance_test(50 * 6);
	run_perfomance_test(50 * 7);
	run_perfomance_test(50 * 8);
	run_perfomance_test(50 * 9);
	run_perfomance_test(50 * 10);

	check(hipDeviceReset());
}
